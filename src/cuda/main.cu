#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <random>
#include <cstdlib>
#include <stack>
#include <unistd.h>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include "particle.cuh"
#include "particle.cu"
#include "vector.cuh"
#include "vector.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

int num_particles;
float particle_size;
Particle* particles;
Particle* device_particles;
hiprandState* states;

// GL functionality
bool initGL(int *argc, char **argv);

// Check for collisions and resolve them
__global__ void checkCollision(Particle* d_particles, int n_particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (int j = i + 1; j < n_particles; j++) {
        if (d_particles[i].collidesWith(d_particles[j])) {
            d_particles[i].resolveCollision(d_particles[j]);
        }
    }
}

// Update the position of the particles and check for wall collisions
__global__ void updateParticles(Particle* d_particles, int n_particles, hiprandState* states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        hiprand_init(i, 0, 0, &states[i]);
        float dx = (float) hiprand_uniform(&states[i]);
        float scaled = (dx / RAND_MAX) * 2 + 2;
        d_particles[i].updatePosition(scaled);
        d_particles[i].wallBounce();
    }
}

// Host function
void display() {

	glClear(GL_COLOR_BUFFER_BIT);

    // Render particles
    for (int i = 0; i < num_particles; i++) {
        particles[i].renderCircle();
    }

    int blockSize = 256;
    int blockCount = (num_particles + blockSize - 1) / blockSize;;

    // Send particle data to device
    hipMemcpy(device_particles, particles, num_particles * sizeof(Particle), hipMemcpyHostToDevice);
    updateParticles<<<blockCount, blockSize>>>(device_particles, num_particles, states);
    checkCollision<<<blockCount, blockSize>>>(device_particles, num_particles);
    // Retrieve particle data from device
    hipMemcpy(particles, device_particles, num_particles * sizeof(Particle), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // FPS counter
    static int frameCount = 0;
    static int lastTime = 0;
    int currentTime = glutGet(GLUT_ELAPSED_TIME);
    frameCount++;

    if (currentTime - lastTime > 1000) {
        char title[80];
        sprintf(title, "Particle Simulator (%d fps) - %d particles", frameCount, num_particles);
        printf("%d\n", frameCount);
        frameCount = 0;
        glutSetWindowTitle(title);
        lastTime = currentTime;
    }

    glutSwapBuffers();
}

void timer( int value )
{
    glutPostRedisplay();
    glutTimerFunc( 16, timer, 0 );
}

bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitWindowSize(800, 800);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutCreateWindow("Particle Simulator");
    glutPositionWindow(950,100);
    glutTimerFunc( 0, timer, 0 );
    glutDisplayFunc(display);

    // Initialize GLEW
    glewExperimental = GL_TRUE;
    GLenum err = glewInit();
    if (err != GLEW_OK) {
        fprintf(stderr, "GLEW initialization failed: %s\n", glewGetErrorString(err));
        return false;
    }

    return true;
}

int main(int argc, char** argv) {
    // Set defaults
    srand(time(NULL));
    num_particles = 5;
    particle_size = 0.1f;
    int opt;

    // Command line options
    while ((opt = getopt(argc, argv, "n:s:")) != -1) {
        switch (opt) {
            case 'n':
                num_particles = strtol(optarg, NULL, 10);
                break;
            case 's':
                particle_size = strtod(optarg, NULL);
                break;
            default:
                fprintf(stderr, "Usage: %s [-n num_particles] [-sp particle_size]\n", argv[0]);
                exit(EXIT_FAILURE);
        }
    }

    particles = (Particle*) calloc(num_particles, sizeof(Particle));

    for (int i = 0; i < num_particles; i++) {
        std::random_device rd;
        std::mt19937 gen(rd());

        // Randomize velocity, position, and mass
        std::uniform_real_distribution<float> dist(-0.0015, 0.0015);
        std::uniform_real_distribution<float> rand(-0.95, 0.95);
        std::uniform_real_distribution<float> mass(1.5, 5.5);

        // Make Particle
        // make random particle velocity        
        float dx = dist(gen) * 6;
        float dy = dist(gen) * 6;
        // make random particle position
        float x = rand(gen);
        float y = rand(gen);
        particles[i] = Particle(Vector(x, y), Vector(dx, dy), mass(gen), particle_size);
    }

    // Init the device particles
    hipMalloc((void**)&device_particles, num_particles * sizeof(Particle));
    hipMalloc((void**)&states, num_particles * sizeof(hiprandState));

    initGL(&argc, argv);
    glutMainLoop();

    // Clean up
    hipDeviceSynchronize();
    hipFree(device_particles);
    hipFree(states);

    return 0;
}