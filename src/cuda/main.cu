#include "hip/hip_runtime.h"
// #include <>
// #include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <random>
#include <cstdlib>
#include <stack>

// #include "helpers/hip/hip_runtime_api.h"
#include <GL/glew.h>
#include <GL/freeglut.h>

// #include <cuda_gl_interop.h>

#include "particle.cuh"
#include "particle.cu"

#include "vector.cuh"
#include "vector.cu"

#define MAX_PARTICLES_PER_NODE 4
#include <math.h>
#define PI 3.14159265f


#define PARTICLE_NUM 10000
#define PARTICLE_SIZE 0.003f

GLuint vertex_buffer;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;
Particle* device_particles;
Particle* particles;

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);


// A cuda kernel
__global__ void checkCollision(Particle* d_particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (int j = i + 1; j < PARTICLE_NUM; j++) {
        if (d_particles[i].collidesWith(d_particles[j])) {
            d_particles[i].resolveCollision(d_particles[j]);
        }
    }
}

void display() {
	glClear(GL_COLOR_BUFFER_BIT);

    for (int i = 0; i < PARTICLE_NUM; i++) {
        particles[i].renderCircle();
        // make a random number
        float dx = (float) rand();
        // scale it to be between 2 and 4
        float scaled = (dx / RAND_MAX) * 2 + 2;
        particles[i].updatePosition(scaled);
        particles[i].wallBounce();

    }

    int blockSize = 256;
    int blockCount = (PARTICLE_NUM + blockSize - 1) / blockSize;

    // Send particle data to device
    hipMemcpy(device_particles, particles, PARTICLE_NUM * sizeof(Particle), hipMemcpyHostToDevice);
    // Do the cuda stuff
    checkCollision<<<blockCount, blockSize>>>(device_particles);
    // Retrieve particle data from device
    hipMemcpy(particles, device_particles, PARTICLE_NUM * sizeof(Particle), hipMemcpyDeviceToHost);

    static int frameCount = 0;
    static int lastTime = 0;
    int currentTime = glutGet(GLUT_ELAPSED_TIME);
    frameCount++;

    if (currentTime - lastTime > 1000) {
        char title[80];
        sprintf(title, "Particle Simulator (%d fps)", frameCount);
        glutSetWindowTitle(title);
        frameCount = 0;
        lastTime = currentTime;
    }

    glutSwapBuffers();
}

void timer( int value )
{
    glutPostRedisplay();
    glutTimerFunc( 16, timer, 0 );
}

bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitWindowSize(800, 800);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutCreateWindow("Particle Simulator");
    glutTimerFunc( 0, timer, 0 );
    glutDisplayFunc(display);


    // Initialize GLEW
    glewExperimental = GL_TRUE;
    GLenum err = glewInit();
    if (err != GLEW_OK) {
        fprintf(stderr, "GLEW initialization failed: %s\n", glewGetErrorString(err));
        return false;
    }

    return true;
}

int main(int argc, char** argv) {
    // const int cuda_device = findCudaDevice(argc, (const char**)argv);
    // hipDeviceProp_t deviceProps;
    // checkCudaErrors(hipGetDeviceProperties(&deviceProps, cuda_device));
    srand(time(NULL));
    particles = (Particle*)malloc(PARTICLE_NUM * sizeof(Particle));

    for (int i = 0; i < PARTICLE_NUM; i++) {
        std::random_device rd;
        std::mt19937 gen(rd());

        std::uniform_real_distribution<float> dist(-0.0015, 0.0015);
        std::uniform_real_distribution<float> rand(-0.95, 0.95);


        // Make Particle -------------
        // make random particle velocity        
        float dx = dist(gen) * 6;
        float dy = dist(gen) * 6;
        // make random particle position
        float x = rand(gen);
        float y = rand(gen);
        particles[i] = Particle(Vector(x, y), Vector(dx, dy), 1, PARTICLE_SIZE);
        // ---------------------------
    }

    // Init the device particles
    hipMalloc((void**)&device_particles, PARTICLE_NUM * sizeof(Particle));

    initGL(&argc, argv);
    //createVBO(&vertex_buffer, &cuda_vbo_resource, 0);

    // VertexBuffer buffer = VertexBuffer(8 * sizeof(float));

    // float vertices[] = {
    //     -0.5f, -0.5f,
    //      0.5f, -0.5f,
    //      0.5f,  0.5f,
    //     -0.5f,  0.5f
    // };
    // buffer.set_data(vertices, 8 * sizeof(float));

    // glEnableVertexAttribArray(0);
    // glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 2, 0);


    // Shader shader = Shader("res/shaders/basic.shader");
    // shader.bind();
    glutMainLoop();

    hipDeviceSynchronize();
    hipFree(device_particles);

    // return 0;
}